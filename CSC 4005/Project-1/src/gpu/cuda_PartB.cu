#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h> // CUDA Header
#include "utils.hpp"

constexpr int FILTER_SIZE = 3;
constexpr float filter[FILTER_SIZE][FILTER_SIZE] = {
    {1.0 / 9, 1.0 / 9, 1.0 / 9},
    {1.0 / 9, 1.0 / 9, 1.0 / 9},
    {1.0 / 9, 1.0 / 9, 1.0 / 9}};

// CUDA kernel function: 3x3 convolution
__global__ void convolution(const unsigned char* input, unsigned char* output,
                            const float* fl, int width, int height,
                            int num_channels)
{
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx > width * height * num_channels) return;

    auto row = idx / num_channels / width;
    auto col = idx / num_channels % width;

    if (row == 0 || row == height - 1 || col == 0 || col == width - 1) return;

    auto color = idx % num_channels;
    int buffer[9];

    // Apply the filter to the image
#pragma unroll(FILTER_SIZE)
    for (auto i = 0; i < FILTER_SIZE; i++)
    {
#pragma unroll(FILTER_SIZE)
        for (auto j = 0; j < FILTER_SIZE; j++)
        {
            auto neighbor_row = row + i - 1;
            auto neighbor_col = col + j - 1;
            auto neighbor_color = color;
            auto neighbor_index = neighbor_row * width * num_channels +
                                  neighbor_col * num_channels + neighbor_color;
            auto filter_value = fl[i * FILTER_SIZE + j];
            auto source_value = input[neighbor_index];
            auto result_value = static_cast<int>(filter_value * source_value);
            buffer[i * FILTER_SIZE + j] = result_value;
        }
    }
    output[idx] = static_cast<unsigned char>(buffer[0] + buffer[1] + buffer[2] +
                                             buffer[3] + buffer[4] + buffer[5] +
                                             buffer[6] + buffer[7] + buffer[8]);
}

int main(int argc, char** argv)
{
    // Verify input arguments format
    if (argc != 3)
    {
        std::cerr << "Invalid argument, should be: ./executable "
                     "/path/to/input/jpeg /path/to/output/jpeg\n";
        return -1;
    }
    // Read from input JPEG file
    const char* input_filepath = argv[1];
    std::cout << "Input file from: " << input_filepath << "\n";
    auto input_jpeg = read_from_jpeg(input_filepath);
    // Allocate memory on host (CPU)
    auto filteredImage =
        new unsigned char[input_jpeg.width * input_jpeg.height *
                          input_jpeg.num_channels];
    // Allocate memory on device (GPU)
    unsigned char* d_input;
    unsigned char* d_output;
    float* d_filter;
    hipMalloc((void**)&d_input, input_jpeg.width * input_jpeg.height *
                                     input_jpeg.num_channels *
                                     sizeof(unsigned char));
    hipMalloc((void**)&d_output, input_jpeg.width * input_jpeg.height *
                                      input_jpeg.num_channels *
                                      sizeof(unsigned char));
    hipMalloc((void**)&d_filter, FILTER_SIZE * FILTER_SIZE * sizeof(float));
    // Copy input data from host to device
    hipMemcpy(d_input, input_jpeg.buffer,
               input_jpeg.width * input_jpeg.height * input_jpeg.num_channels *
                   sizeof(unsigned char),
               hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, FILTER_SIZE * FILTER_SIZE * sizeof(float),
               hipMemcpyHostToDevice);
    // Computation: Filter
    hipEvent_t start, stop;
    float gpuDuration;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Calculate the number of blocks and threads
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum
                     // occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, convolution);
    gridSize = (input_jpeg.width * input_jpeg.height * input_jpeg.num_channels +
                blockSize - 1) /
               blockSize;

    // GPU start time
    hipEventRecord(start, 0);
    // Call kernel function
    convolution<<<gridSize, blockSize>>>(d_input, d_output, d_filter,
                                         input_jpeg.width, input_jpeg.height,
                                         input_jpeg.num_channels);
    // GPU stop time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // Print the GPU execution time
    hipEventElapsedTime(&gpuDuration, start, stop);
    // Copy output data from device to host
    hipMemcpy(filteredImage, d_output,
               input_jpeg.width * input_jpeg.height * input_jpeg.num_channels *
                   sizeof(unsigned char),
               hipMemcpyDeviceToHost);
    // Write to output JPEG file
    const char* output_filepath = argv[2];
    std::cout << "Output file to: " << output_filepath << "\n";
    JPEGMeta output_jpeg{filteredImage, input_jpeg.width, input_jpeg.height,
                         input_jpeg.num_channels, input_jpeg.color_space};
    if (write_to_jpeg(output_jpeg, output_filepath))
    {
        std::cerr << "Failed to write output JPEG\n";
        return -1;
    }
    // Free memory on host (CPU)
    delete[] filteredImage;
    delete[] input_jpeg.buffer;
    // Free memory on device (GPU)
    hipFree(d_input);
    hipFree(d_output);

    std::cout << "Smoothing Complete!" << std::endl;
    std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds"
              << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}